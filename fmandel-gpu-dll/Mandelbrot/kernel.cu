#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"
#include <jni.h>
#include <fstream>
#include <vector>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void mandelbrot_kernel(unsigned char* image, int width, int height, double x_min, double x_max, double y_min, double y_max, int max_iter, double zoom_x, double zoom_y, int zoom_level) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px < width && py < height) {

        double zoomed_x_min = zoom_x - (x_max - x_min) / (2.0f * zoom_level);
        double zoomed_x_max = zoom_x + (x_max - x_min) / (2.0f * zoom_level);
        double zoomed_y_min = zoom_y - (y_max - y_min) / (2.0f * zoom_level);
        double zoomed_y_max = zoom_y + (y_max - y_min) / (2.0f * zoom_level);

        double x0 = zoomed_x_min + (px / (double)width) * (zoomed_x_max - zoomed_x_min);
        double y0 = zoomed_y_min + (py / (double)height) * (zoomed_y_max - zoomed_y_min);
        double x = 0.0;
        double y = 0.0;
        int iteration = 0;

        while (x * x + y * y <= 4.0 && iteration < max_iter) {
            double xtemp = x * x - y * y + x0;
            y = 2 * x * y + y0;
            x = xtemp;
            iteration++;
        }
        int pixelIndex = py * width + px * 3;
        
        double t = double(iteration) / max_iter;
        image[pixelIndex] = (unsigned char)(9 * (1 - t) * t * t * t * 255);
        image[pixelIndex + 1] = (unsigned char)(15 * (1 - t) * (1 - t) * t * t * 255);
        image[pixelIndex + 2] = (unsigned char)(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);
        
    }
}

extern "C" {

    __declspec(dllexport) unsigned char* allocate_image(int width, int height) {
        unsigned char* image;
        hipMallocManaged(&image, width * height * 3);
        return image;
    }

    __declspec(dllexport)  void free_image(unsigned char* image) {
        hipFree(image);
    }

    __declspec(dllexport)  void generate_mandelbrot(unsigned char* image, int width, int height,
        double x_center, double y_center,
        double x_min, double x_max,
        double y_min, double y_max,
        int max_iter, int zoom_steps) {
        dim3 threads_per_block(16, 16);
        dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x,
            (height + threads_per_block.y - 1) / threads_per_block.y);

        mandelbrot_kernel <<<num_blocks, threads_per_block>>> (image, width, height, x_min, x_max, y_min, y_max, max_iter, x_center, y_center, zoom_steps);
        hipDeviceSynchronize();
    }

    __declspec(dllexport)  void save_image(const char* filename, unsigned char* image, int width, int height) {
        std::ofstream ofs(filename, std::ios::binary);
        ofs << "P6\n" << width << " " << height << "\n255\n";
        ofs.write(reinterpret_cast<char*>(image), width * height * 3);
        ofs.close();
    }

}

//extern "C" JNIEXPORT jobject JNICALL Java_kt_MandelbrotLibrary_allocateImage(JNIEnv* env, jobject obj, jint width, jint height) {
//    unsigned char* image = allocate_image(width, height);
//    return env->NewDirectByteBuffer(image, width * height * 3);  // Create a direct ByteBuffer from native memory
//}
//
//extern "C" JNIEXPORT void JNICALL Java_kt_MandelbrotLibrary_freeImage(JNIEnv* env, jobject obj, jobject buffer) {
//    unsigned char* image = (unsigned char*)env->GetDirectBufferAddress(buffer);  // Get the pointer from ByteBuffer
//    free_image(image);  // Free the native memory
//}
//
//extern "C" JNIEXPORT void JNICALL Java_kt_MandelbrotLibrary_generateMandelbrot(JNIEnv* env, jobject obj,
//    jobject imageBuffer, jint width, jint height,
//    jdouble xCenter, jdouble yCenter,
//    jdouble xMin, jdouble xMax,
//    jdouble yMin, jdouble yMax,
//    jint maxIter, jint zoomSteps) {
//    // Get the pointer from the ByteBuffer
//    unsigned char* image = (unsigned char*)env->GetDirectBufferAddress(imageBuffer);
//
//    // Check if the pointer is valid
//    if (image == nullptr) {
//        printf("Error: GetDirectBufferAddress returned NULL\n");
//        return;
//    }
//
//    // Call the CUDA function to generate the Mandelbrot set
//    generate_mandelbrot(image, width, height, xCenter, yCenter, xMin, xMax, yMin, yMax, maxIter, zoomSteps);
//}
//
//extern "C" JNIEXPORT void JNICALL Java_kt_MandelbrotLibrary_saveImage(JNIEnv* env, jobject obj,
//    jstring filename, jobject imageBuffer,
//    jint width, jint height) {
//    // Get the pointer from the ByteBuffer
//    unsigned char* image = (unsigned char*)env->GetDirectBufferAddress(imageBuffer);
//
//    // Get the filename from the jstring
//    const char* file = env->GetStringUTFChars(filename, nullptr);
//
//    // Call the CUDA function to save the image
//    save_image(file, image, width, height);
//
//    // Release the filename memory
//    env->ReleaseStringUTFChars(filename, file);
//}
void write_to_memory(void* context, void* data, int size) {
    std::vector<unsigned char>* buffer = static_cast<std::vector<unsigned char>*>(context);
    buffer->insert(buffer->end(), (unsigned char*)data, (unsigned char*)data + size);
}


extern "C" JNIEXPORT jbyteArray JNICALL Java_com_jmvsta_fmandelbackend_MandelbrotLibrary_makePicture(JNIEnv* env, jobject obj,
    jint width, jint height,
    jdouble xCenter, jdouble yCenter,
    jdouble xMin, jdouble xMax,
    jdouble yMin, jdouble yMax,
    jint maxIter, jint zoomSteps) {

    unsigned char* image = allocate_image(width, height);
    
    generate_mandelbrot(image, width, height, xCenter, yCenter, xMin, xMax, yMin, yMax, maxIter, zoomSteps);
    std::vector<unsigned char> jpegData;
    stbi_write_jpg_to_func(write_to_memory, &jpegData, width, height, 3, image, 95);

    
    free_image(image);
    if (jpegData.size() > static_cast<size_t>(std::numeric_limits<jsize>::max())) {
        return nullptr;
    }

    jsize arraySize = static_cast<jsize>(jpegData.size());


    jbyteArray result = env->NewByteArray(arraySize);
    env->SetByteArrayRegion(result, 0, arraySize, reinterpret_cast<jbyte*>(jpegData.data()));
    return result;
}
